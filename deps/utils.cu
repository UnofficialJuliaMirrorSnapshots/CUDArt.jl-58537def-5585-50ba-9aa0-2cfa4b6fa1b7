
#include <hip/hip_runtime.h>
#include <stdint.h>
//#include <stdio.h>

template <typename T>
__device__ void fill_contiguous(T *data, size_t len, T val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = idx; i < len; i += gridDim.x * blockDim.x) {
        data[i] = val;
    }
}

// Note that pitch must be expressed in elements, not bytes!
template <typename T>
__device__ void fill_pitched(T *data, size_t width, size_t height, size_t depth, size_t pitch, T val) {
    int idxx = blockIdx.x * blockDim.x + threadIdx.x;
    int idxy = blockIdx.y * blockDim.y + threadIdx.y;
    int idxz = blockIdx.z * blockDim.z + threadIdx.z;

    for (int iz = idxz; iz < depth; iz += gridDim.z * blockDim.z) {
        int offsetz = iz;
        for (int iy = idxy; iy < height; iy += gridDim.y * blockDim.y) {
            int offsety = height*offsetz + iy;
            for (int ix = idxx; ix < width; ix += gridDim.x * blockDim.x) {
                data[pitch*offsety + ix] = val;
            }
        }
    }
}

extern "C"
{
    void __global__ fill_contiguous_double(double *data, size_t len, double val) {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_float(float *data, size_t len, float val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int64(int64_t *data, size_t len, int64_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint64(uint64_t *data, size_t len, uint64_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int32(int32_t *data, size_t len, int32_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint32(uint32_t *data, size_t len, uint32_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int16(int16_t *data, size_t len, int16_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint16(uint16_t *data, size_t len, uint16_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int8(int8_t *data, size_t len, int8_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint8(uint8_t *data, size_t len, uint8_t val)  {fill_contiguous(data, len, val);}

    void __global__ fill_pitched_double(double *data, size_t width, size_t height, size_t depth, size_t pitch, double val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_float(float *data, size_t width, size_t height, size_t depth, size_t pitch, float val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int64(int64_t *data, size_t width, size_t height, size_t depth, size_t pitch, int64_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint64(uint64_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint64_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int32(int32_t *data, size_t width, size_t height, size_t depth, size_t pitch, int32_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint32(uint32_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint32_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int16(int16_t *data, size_t width, size_t height, size_t depth, size_t pitch, int16_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint16(uint16_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint16_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int8(int8_t *data, size_t width, size_t height, size_t depth, size_t pitch, int8_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint8(uint8_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint8_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
}

// For implementing sleep, from http://stackoverflow.com/questions/11217117/equivalent-of-usleep-in-cuda-kernel
extern "C" {
// __global__ void clock_block(int64_t *d_o, int64_t clock_count)
__global__ void clock_block(int64_t clock_count)
{
    int64_t start_clock = clock64();
    int64_t clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock64() - start_clock;
    }
//     d_o[0] = (int64_t) clock_offset;
}
}

/*// Debugging
int main()
{
  size_t width = 5;
  size_t height = 3;
  size_t depth = 1;
  size_t pitch = sizeof(double)*width;
  cudaSetDevice(0);
  cudaExtent extent = make_cudaExtent(pitch, height, depth);
  cudaPitchedPtr devpp;
  cudaMalloc3D(&devpp, extent);
  pitch = devpp.pitch;
  fill_pitched_double<<<32,16>>>((double *) devpp.ptr, width, height, depth, pitch/sizeof(double), (double) 8.4);
  cudaDeviceSynchronize();
  double *host = new double[width*height];
  cudaPos dstpos = make_cudaPos(0, 0, 0);
  cudaPos srcpos = make_cudaPos(0, 0, 0);
  cudaPitchedPtr hostpp = make_cudaPitchedPtr(host, sizeof(double)*width, width, height);
  cudaMemcpy3DParms mcpp = {0};
  mcpp.srcPos = srcpos;
  mcpp.dstPos = dstpos;
  mcpp.srcPtr = devpp;
  mcpp.dstPtr = hostpp;
  mcpp.extent = extent;
  mcpp.kind = cudaMemcpyDeviceToHost;
  cudaMemcpy3D(&mcpp);
  cudaDeviceSynchronize();
  for (int i = 0; i < width*height; i++)
    printf("%g\n", host[i]);
  cudaDeviceReset();
}*/