
#include <hip/hip_runtime.h>
// a simple CUDA kernel to add two vectors

extern "C" 
{

__global__ void vadd(const float *a, const float *b, float *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}

} // extern "C"

